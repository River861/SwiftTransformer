#include "hip/hip_runtime.h"
#include "fused_decoding_stage_attention.h"

#include <cassert>
#include <cstdio>
#include <cstdint>

#include "util/cuda_utils.h"

namespace st::kernel {

#define WARP_SIZE 32

// Tuneable parameters
constexpr int64_t DEFAULT_THREAD_BLOCK_SIZE = 256;

/*
	# fusedDecodingStageAttentionKernel

	## Overview

	This kernel saves K/V cache of the latest token and performs batched & fused decoding stage attention.
	
	Recall how the array of token (the `input` array) looks like in `attention()` in `layer/attention.cc`:
	| Prompt or last token | Prompt or last token | ... | Prompt or last token |

	If a request is in the context stage, then "Prompt or last token" contains (the length of) the prompt
	number of tokens. Otherwise, if a request is in the decoding stage, then it contains the last token.

	This kernel only focuses on requests that are in the latter case, i.e. the decoding stage. It
	takes input tokens, k cache and v cache, calculate softmax(qK^T)V (Here q is a vector of length
	num_heads*head_dim), and store the result in `result`

	## Parameters

	Since not every request is going to be processed, we need to pass an array `ith_decoding_request_index`, 
	which contains the index of the request in the decoding stage. For example, if the input is
	| Context tokens | Context tokens | Decoding token | Decoding token | Context tokens | Decoding token |,
	then `ith_decoding_request_index` should be [2, 3, 5].

	## Algorithm & Implementation Details

	Similar to FlashAttention's but the number of query vectors = 1
*/

template<
	typename T,
	int64_t Q_HEADS_PER_KV_HEAD,
	int64_t HEAD_DIM,
	int64_t BLOCK_SIZE,
	int64_t THREAD_BLOCK_SIZE
> __global__ void fusedDecodingStageAttentionKernel(
	// The output
	T* __restrict__ result,			// [num_tokens, num_q_heads, head_dim]

	// QKVs
	const T* __restrict__ qkvs,	// [num_tokens, num_q_heads+2*num_kv_heads, head_dim]
	T* __restrict__ k_cache_offseted,		// The OFFSETed k_cache.
								// The shape of k_cache is [num_blocks, num_layers, num_kv_heads, block_size, head_dim]
								// This k_cache_offseted is real k_cache + layer_id*num_kv_heads*block_size*head_dim
								// So we does not need another register for storing layer_id
	T* __restrict__ v_cache_offseted,		// [num_blocks, num_layers, num_kv_heads, block_size, head_dim]

	// Other inputs
	const float qk_scale,				// 1/sqrt(head_dim)
	const int64_t* __restrict__ block_table,	// [num_reqs, max_num_block_per_seq]
	const int64_t* __restrict__ input_lens,		// [num_reqs]. Here input_lens DOES NOT INCLUDE the latest token!
	const int64_t* __restrict__ ith_decoding_req_req_index,	// [num_decoding_reqs]
	const int64_t* __restrict__ ith_decoding_req_token_index,	// [num_decoding_reqs]
	const int64_t max_num_block_per_seq,
	const int64_t num_layers
) {
	constexpr int64_t NUM_THREAD_PER_KEY = WARP_SIZE / BLOCK_SIZE;	// The size of the thread group
	constexpr int64_t THREAD_GROUP_SIZE = NUM_THREAD_PER_KEY;		// Just an alias
	constexpr int64_t NUM_WARPS = THREAD_BLOCK_SIZE / WARP_SIZE;
	constexpr int64_t NUM_ELEM_PER_THREAD = (HEAD_DIM/2) / NUM_THREAD_PER_KEY;
	typedef std::conditional_t<std::is_same<T, half>::value, half2, float2> T2;

	const int64_t q_head_id = blockIdx.x;		// Grid: num_q_heads x num_decoding_reqs
	const int64_t kv_head_id = q_head_id / Q_HEADS_PER_KV_HEAD;
	const int64_t num_q_heads = gridDim.x;	// TODO Pass it as a template parameter to save a register
	const int64_t num_kv_heads = num_q_heads / Q_HEADS_PER_KV_HEAD;

	const int64_t req_index = ith_decoding_req_req_index[blockIdx.y];
	const int64_t token_index = ith_decoding_req_token_index[blockIdx.y];
	const int64_t input_len = input_lens[req_index];	// Here input_lens DOES NOT INCLUDE the latest token!
	const int64_t num_blocks = (input_len+1 + BLOCK_SIZE - 1) / BLOCK_SIZE;

	const int64_t warp_id = threadIdx.x / WARP_SIZE;	// Which warp we are in
	const int64_t lane_id = threadIdx.x % WARP_SIZE;
	const int64_t thread_group_id = lane_id / NUM_THREAD_PER_KEY;	// Which thread group we are in, i.e. which column in K^T we are responsible for
	const int64_t thread_id_in_group = lane_id % NUM_THREAD_PER_KEY;	// Which thread we are in the thread group

	extern __shared__ float shared_mem[];
	float* attn_score = shared_mem; // [\lceil max_input_len/BLOCK_SIZE \rceil * BLOCK_SIZE]
	T2* qkv_reduction_wksp = (T2*)shared_mem;	// [NUM_WARPS, WARP_SIZE]
	__shared__ float reduction_wksp[32];	// Workspace for reduction. Here 32 >= NUM_WARPS

	// Step 0: Save the KV cache
	if (threadIdx.x < HEAD_DIM) {
		int64_t kvcache_index = INDEX_5D(
			0, num_layers, num_kv_heads, BLOCK_SIZE, HEAD_DIM,
			block_table[INDEX_2D(0, max_num_block_per_seq, req_index, input_len/BLOCK_SIZE)],
			0, kv_head_id, input_len%BLOCK_SIZE, threadIdx.x
		);
		k_cache_offseted[kvcache_index] = qkvs[INDEX_3D(0, num_q_heads+2*num_kv_heads, HEAD_DIM, token_index, num_q_heads + kv_head_id, threadIdx.x)];
		v_cache_offseted[kvcache_index] = qkvs[INDEX_3D(0, num_q_heads+2*num_kv_heads, HEAD_DIM, token_index, num_q_heads + num_kv_heads + kv_head_id, threadIdx.x)];
	}
	__syncthreads();	// Since we are going to use k_cache and v_cache later

	// Step 1: Load q into registers
	// 
	// We do this since we must multiply q with every column in K^T, and we can save a lot of
	// global memory access by doing this.
	// 
	// To leverage the memory coalescing, the i-th thread in the thread group
	// is responsible for q[i], q[i+THREAD_GROUP_SIZE], and so on.
	T2 q_cache[NUM_ELEM_PER_THREAD];
	#pragma unroll
	for (int64_t i = 0; i < NUM_ELEM_PER_THREAD; i++) {
		q_cache[i] = ((const T2 *)qkvs)[INDEX_3D(0, num_q_heads+2*num_kv_heads, HEAD_DIM/2, token_index, q_head_id, thread_id_in_group + i*THREAD_GROUP_SIZE)];
	}

	// Variables for softmax-ing
	float max_qki = -__FLT_MAX__;

	// Iterate over all blocks
	for (int64_t block_idx = warp_id; block_idx < num_blocks; block_idx += NUM_WARPS) {
		const int64_t block_index = block_table[req_index*max_num_block_per_seq + block_idx];
		const T2* k_block = (const T2*)(k_cache_offseted + (block_index*num_layers*num_kv_heads + kv_head_id)*BLOCK_SIZE*HEAD_DIM);
		const int64_t token_idx = block_idx*BLOCK_SIZE + thread_group_id;

		// Step 2: Calculate qkij
		float qkij = 0;
		#pragma unroll
		for (int64_t i = 0; i < NUM_ELEM_PER_THREAD; ++i) {
			const T2 q_elem = q_cache[i];
			const T2 k_elem = k_block[INDEX_2D(0, HEAD_DIM/2, thread_group_id, thread_id_in_group + i*NUM_THREAD_PER_KEY)];
			qkij += (float)(q_elem.x * k_elem.x + q_elem.y * k_elem.y);
		}

		// Step 3: Reduce qkij to get qki
		float qki = qkij;
		#pragma unroll
		for (int64_t mask = THREAD_GROUP_SIZE/2; mask; mask >>= 1) {
			qki += __shfl_xor_sync(0xffffffff, qki, mask);
		}
		// Now all threads with thread_id_in_group == 0 has the correct value of qki
		if (thread_id_in_group == 0) {
			qki = token_idx <= input_len ? qki*qk_scale : -__FLT_MAX__;
			max_qki = fmaxf(max_qki, qki);
			attn_score[token_idx] = qki;
		}
	}

	// Step 4: Perform reduction on max_qki within each warp
	#pragma unroll
	for (int mask = WARP_SIZE/2; mask >= THREAD_GROUP_SIZE; mask >>= 1) {
		max_qki = fmaxf(max_qki, __shfl_xor_sync(0xffffffff, max_qki, mask));
	}
	// Now all threads with lane == 0 has max_qki = max(qki | i is in the same warp)
	if (lane_id == 0) {
		reduction_wksp[warp_id] = max_qki;
	}
	__syncthreads();

	// Step 5: Perform reduction on max_qki within the whole thread group
	if (warp_id == 0) {
		max_qki = lane_id < NUM_WARPS ? reduction_wksp[lane_id] : -__FLT_MAX__;
		#pragma unroll
		for (int mask = NUM_WARPS/2; mask; mask >>= 1) {
			max_qki = fmaxf(max_qki, __shfl_xor_sync(0xffffffff, max_qki, mask));
		}
		// Now thread #0 has the correct max_qki
		if (lane_id == 0) {
			reduction_wksp[0] = max_qki;
		}
	}
	__syncthreads();

	// Step 6: Get the sum of exp(qki - max_qki)
	max_qki = reduction_wksp[0];
	float sum_exp_qki = 0;
	#pragma unroll
	for (int i = threadIdx.x; i < num_blocks*BLOCK_SIZE; i += THREAD_BLOCK_SIZE) {
		float val = __expf(attn_score[i] - max_qki);
		sum_exp_qki += val;
		attn_score[i] = val;
	}
	__syncthreads();

	// Perform reduction within warp
	#pragma unroll
	for (int mask = WARP_SIZE/2; mask; mask >>= 1) {
		sum_exp_qki += __shfl_xor_sync(0xffffffff, sum_exp_qki, mask);
	}
	if (lane_id == 0) {
		reduction_wksp[warp_id] = sum_exp_qki;
	}
	__syncthreads();

	// Perform reduction within thread group
	if (warp_id == 0) {
		sum_exp_qki = lane_id < NUM_WARPS ? reduction_wksp[lane_id] : 0;
		#pragma unroll
		for (int mask = NUM_WARPS/2; mask; mask >>= 1) {
			sum_exp_qki += __shfl_xor_sync(0xffffffff, sum_exp_qki, mask);
		}
		if (lane_id == 0) {
			reduction_wksp[0] = 1.0f / (sum_exp_qki + 1e-6f);
		}
	}
	__syncthreads();

	// Step 7: Calculate softmax
	float softmax_denorm = reduction_wksp[0];
	#pragma unroll
	for (int token_index = threadIdx.x; token_index <= input_len; token_index += THREAD_BLOCK_SIZE) {
		attn_score[token_index] *= softmax_denorm;
	}
	__syncthreads();

	// Step 8: calculate attn_score * V
	constexpr int64_t NUM_COL_PER_THREAD = (HEAD_DIM/2 + WARP_SIZE - 1) / WARP_SIZE;
	T2 acc[NUM_COL_PER_THREAD];
	#pragma unroll
	for (int i = 0; i < NUM_COL_PER_THREAD; ++i) {
		acc[i].x = acc[i].y = 0;
	}

	// Iterate over all blocks
	#pragma unroll
	for (int block_idx = warp_id; block_idx < num_blocks; block_idx += NUM_WARPS) {
		const int64_t block_index = block_table[req_index*max_num_block_per_seq + block_idx];
		const T2* v_block = (const T2*)(v_cache_offseted + (block_index*num_layers*num_kv_heads + kv_head_id)*BLOCK_SIZE*HEAD_DIM);
		const int64_t token_idx = block_idx*BLOCK_SIZE;

		#pragma unroll
		for (int col = lane_id; col < HEAD_DIM/2; col += WARP_SIZE) {
			T2 acc_elem = acc[col/WARP_SIZE];
			T sum_x = 0, sum_y = 0;
			#pragma unroll
			for (int i = 0; i < BLOCK_SIZE; i += 1) {
				T attn_score_elem = (T)attn_score[token_idx + i];
				T2 v_elem = v_block[INDEX_2D(BLOCK_SIZE, HEAD_DIM/2, i, col)];
				sum_x += attn_score_elem * v_elem.x;
				sum_y += attn_score_elem * v_elem.y;
			}
			acc[col/WARP_SIZE] = {acc_elem.x + sum_x, acc_elem.y + sum_y};
		}
	}
	__syncthreads();

	// Reduce accs among threads with the same lane_id
	#pragma unroll
	for (int i = 0; i < NUM_COL_PER_THREAD; ++i) {
		// In the "Iterate over all blocks" above, each thread is responsible
		// for column `lane_id`, `lane_id+WARP_SIZE`, `lane_id+2*WARP_SIZE`, ...

		// Now we focus on col = i*WARP_SIZE + lane_id
		// Copy the cols to the shared memory
		{
			int col = i*WARP_SIZE + lane_id;
			if (col < HEAD_DIM/2) {
				qkv_reduction_wksp[warp_id*WARP_SIZE + lane_id] = acc[i];
			}
			__syncthreads();
		}

		// Now our task is to, for every i in 0..WARP_SIZE-1,
		// summing up qkv_reduction_wksp[0][i], qkv_reduction_wksp[1][i], ... qkv_reduction_wksp[NUM_WARPS-1][i]
		// The i-th thread calculates the sum above for column threadIdx.x
		{
			const int thread_id = threadIdx.x;
			if (thread_id < WARP_SIZE && i*WARP_SIZE + thread_id < HEAD_DIM/2) {
				T sum_x = 0, sum_y = 0;
				#pragma unroll
				for (int j = 0; j < NUM_WARPS; ++j) {
					const T2 elem = qkv_reduction_wksp[j*WARP_SIZE + thread_id];
					sum_x += elem.x;
					sum_y += elem.y;
				}
				((T2*)result)[INDEX_3D(0, num_q_heads, HEAD_DIM/2, token_index, q_head_id, i*WARP_SIZE + thread_id)] = {sum_x, sum_y};
			}
			__syncthreads();
		}
	}
}

#define LAUNCH_DECODING_STAGE_ATTENTION_KERNEL(T, Q_HEADS_PER_KV_HEAD, HEAD_DIM, BLOCK_SIZE) \
	fusedDecodingStageAttentionKernel<T, Q_HEADS_PER_KV_HEAD, HEAD_DIM, BLOCK_SIZE, DEFAULT_THREAD_BLOCK_SIZE><<<grid_dim, DEFAULT_THREAD_BLOCK_SIZE, shared_mem_size>>>( \
		result, qkvs, k_cache_offseted, v_cache_offseted, scale, block_table, input_lens, ith_decoding_req_req_index, ith_decoding_req_token_index, max_num_block_per_seq, num_layers \
	)

#define FUSED_DECODING_STAGE_ATTENTION_DISPATCH_BLOCK_SIZE(T, Q_HEADS_PER_KV_HEAD, HEAD_DIM) \
	switch (block_size) { \
		case 1: LAUNCH_DECODING_STAGE_ATTENTION_KERNEL(T, Q_HEADS_PER_KV_HEAD, HEAD_DIM, 1); break; \
		case 2: LAUNCH_DECODING_STAGE_ATTENTION_KERNEL(T, Q_HEADS_PER_KV_HEAD, HEAD_DIM, 2); break; \
		case 4: LAUNCH_DECODING_STAGE_ATTENTION_KERNEL(T, Q_HEADS_PER_KV_HEAD, HEAD_DIM, 4); break; \
		case 8: LAUNCH_DECODING_STAGE_ATTENTION_KERNEL(T, Q_HEADS_PER_KV_HEAD, HEAD_DIM, 8); break; \
		case 16: LAUNCH_DECODING_STAGE_ATTENTION_KERNEL(T, Q_HEADS_PER_KV_HEAD, HEAD_DIM, 16); break; \
		case 32: LAUNCH_DECODING_STAGE_ATTENTION_KERNEL(T, Q_HEADS_PER_KV_HEAD, HEAD_DIM, 32); break; \
		default: fprintf(stderr, "Unsupported block_size: %ld\n", block_size); assert(0); \
	}

#define FUSED_DECODING_STAGE_ATTENTION_DISPATCH_HEAD_DIM(T, Q_HEADS_PER_KV_HEAD) \
	switch (head_dim) {	\
		case 64: FUSED_DECODING_STAGE_ATTENTION_DISPATCH_BLOCK_SIZE(T, Q_HEADS_PER_KV_HEAD, 64); break;	\
		case 80: FUSED_DECODING_STAGE_ATTENTION_DISPATCH_BLOCK_SIZE(T, Q_HEADS_PER_KV_HEAD, 80); break;	\
		case 128: FUSED_DECODING_STAGE_ATTENTION_DISPATCH_BLOCK_SIZE(T, Q_HEADS_PER_KV_HEAD, 128); break;	\
		default: fprintf(stderr, "Unsupported head_dim: %ld\n", head_dim); assert(0);			\
	}

#define FUSED_DECODING_STAGE_ATTENTION_DISPATCH_Q_HEADS_PER_KV_HEAD(T) \
	switch (q_heads_per_kv_head) {	\
		case 1: FUSED_DECODING_STAGE_ATTENTION_DISPATCH_HEAD_DIM(T, 1); break;	\
		case 2: FUSED_DECODING_STAGE_ATTENTION_DISPATCH_HEAD_DIM(T, 2); break;	\
		case 4: FUSED_DECODING_STAGE_ATTENTION_DISPATCH_HEAD_DIM(T, 4); break;	\
		case 8: FUSED_DECODING_STAGE_ATTENTION_DISPATCH_HEAD_DIM(T, 8); break;	\
		default: fprintf(stderr, "Unsupported q_heads_per_kv_head: %ld\n", q_heads_per_kv_head); assert(0);	\
	}

template<typename T>
void fusedDecodingStageAttention(
	T* __restrict__ result,
	const T* __restrict__ qkvs,
	T* k_cache,
	T* v_cache,
	const float scale,
	const int64_t* __restrict__ block_table,
	const int64_t* __restrict__ input_lens,
	const int64_t num_decoding_reqs,
	const int64_t* __restrict__ ith_decoding_req_req_index,
	const int64_t* __restrict__ ith_decoding_req_token_index,
	const int64_t max_decoding_req_len,
	const int64_t num_layers,
	const int64_t num_q_heads,
	const int64_t num_kv_heads,
	const int64_t head_dim,
	const int64_t layer_id,
	const int64_t block_size,
	const int64_t max_num_block_per_seq
) {
	#ifdef DEBUG
		assert (block_size <= WARP_SIZE);
		assert (DEFAULT_THREAD_BLOCK_SIZE >= head_dim);
	#endif
	if (num_q_heads == num_kv_heads) {
		fusedDecodingStageAttentionMHA(
			result,
			qkvs,
			k_cache,
			v_cache,
			scale,
			block_table,
			input_lens,
			num_decoding_reqs,
			ith_decoding_req_req_index,
			ith_decoding_req_token_index,
			max_decoding_req_len,
			num_layers,
			num_q_heads,
			head_dim,
			layer_id,
			block_size,
			max_num_block_per_seq
		);
		return;
	}
	int64_t q_heads_per_kv_head = num_q_heads / num_kv_heads;
	T* k_cache_offseted = k_cache + layer_id * num_kv_heads * block_size * head_dim;
	T* v_cache_offseted = v_cache + layer_id * num_kv_heads * block_size * head_dim;
	dim3 grid_dim(num_q_heads, num_decoding_reqs);
	int shared_mem_size = std::max(((max_decoding_req_len+1 + block_size-1) / block_size) * block_size * sizeof(float), DEFAULT_THREAD_BLOCK_SIZE*2*sizeof(T));
	FUSED_DECODING_STAGE_ATTENTION_DISPATCH_Q_HEADS_PER_KV_HEAD(T);
}

#define INSTANTIATE_FUSED_DECODING_STAGE_ATTENTION(T) \
	template void fusedDecodingStageAttention( \
		T* __restrict__, \
		const T* __restrict__, \
		T* __restrict__, \
		T* __restrict__, \
		const float, \
		const int64_t* __restrict__, \
		const int64_t* __restrict__, \
		const int64_t, \
		const int64_t* __restrict__, \
		const int64_t* __restrict__, \
		const int64_t, \
		const int64_t, \
		const int64_t, \
		const int64_t, \
		const int64_t, \
		const int64_t, \
		const int64_t, \
		const int64_t \
	);

INSTANTIATE_FUSED_DECODING_STAGE_ATTENTION(float)
INSTANTIATE_FUSED_DECODING_STAGE_ATTENTION(half)

} // namespace st::kernel