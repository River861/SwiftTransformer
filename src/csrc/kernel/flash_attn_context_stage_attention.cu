#include "hip/hip_runtime.h"
#include "flash_attn_context_stage_attention.h"

#include <cassert>
#include <iostream>

#include <pybind11/embed.h>

#include "util/cuda_utils.h"
#include "util/debug_utils.h"

/******************************************************************************
 * Copyright (c) 2024, Tri Dao.
 ******************************************************************************/

// Include these 2 headers instead of torch/extension.h since we don't need all of the torch headers.
#include <torch/python.h>
#include <torch/nn/functional.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <ATen/cuda/CUDAGeneratorImpl.h>  // For at::Generator and at::PhiloxCudaState

#include <cutlass/numeric_types.h>

#include "hardware_info.h"
#include "flash.h"
#include "static_switch.h"

#define CHECK_DEVICE(x) TORCH_CHECK(x.is_cuda(), #x " must be on CUDA")
#define CHECK_SHAPE(x, ...) TORCH_CHECK(x.sizes() == torch::IntArrayRef({__VA_ARGS__}), #x " must have shape (" #__VA_ARGS__ ")")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")

namespace flash_attention {

void set_params_fprop(Flash_fwd_params &params,
                      // sizes
                      const size_t b,
                      const size_t seqlen_q,
                      const size_t seqlen_k,
                      const size_t seqlen_q_rounded,
                      const size_t seqlen_k_rounded,
                      const size_t h,
                      const size_t h_k,
                      const size_t d,
                      const size_t d_rounded,
                      // device pointers
                      const at::Tensor q,
                      const at::Tensor k,
                      const at::Tensor v,
                      at::Tensor out,
                      void *cu_seqlens_q_d,
                      void *cu_seqlens_k_d,
                      void *seqused_k,
                      void *p_d,
                      void *softmax_lse_d,
                      float p_dropout,
                      float softmax_scale,
                      int window_size_left,
                      int window_size_right,
                      const float softcap,
                      bool seqlenq_ngroups_swapped=false,
                      const bool unpadded_lse=false) {

    // Reset the parameters
    params = {};

    params.is_bf16 = q.dtype() == torch::kBFloat16;

    // Set the pointers and strides.
    params.q_ptr = q.data_ptr();
    params.k_ptr = k.data_ptr();
    params.v_ptr = v.data_ptr();
    // All stride are in elements, not bytes.
    params.q_row_stride = q.stride(-3);
    params.k_row_stride = k.stride(-3);
    params.v_row_stride = v.stride(-3);
    params.q_head_stride = q.stride(-2);
    params.k_head_stride = k.stride(-2);
    params.v_head_stride = v.stride(-2);
    params.o_ptr = out.data_ptr();
    params.o_row_stride = out.stride(-3);
    params.o_head_stride = out.stride(-2);

    if (cu_seqlens_q_d == nullptr) {
        params.q_batch_stride = q.stride(0);
        params.k_batch_stride = k.stride(0);
        params.v_batch_stride = v.stride(0);
        params.o_batch_stride = out.stride(0);
        if (seqlenq_ngroups_swapped) {
             params.q_batch_stride *= seqlen_q;
             params.o_batch_stride *= seqlen_q;
        }
    }

    params.cu_seqlens_q = static_cast<int *>(cu_seqlens_q_d);
    params.cu_seqlens_k = static_cast<int *>(cu_seqlens_k_d);
    params.seqused_k = static_cast<int *>(seqused_k);

    // P = softmax(QK^T)
    params.p_ptr = p_d;

    // Softmax sum
    params.softmax_lse_ptr = softmax_lse_d;

    // Set the dimensions.
    params.b = b;
    params.h = h;
    params.h_k = h_k;
    params.h_h_k_ratio = h / h_k;
    params.seqlen_q = seqlen_q;
    params.seqlen_k = seqlen_k;
    params.seqlen_q_rounded = seqlen_q_rounded;
    params.seqlen_k_rounded = seqlen_k_rounded;
    params.d = d;
    params.d_rounded = d_rounded;

    // Set the different scale values.
    #ifdef FLASHATTENTION_DISABLE_SOFTCAP
        TORCH_CHECK(softcap <= 0.0, "This flash attention build does not support softcap.");
    #endif
    if (softcap > 0.0) {
        params.softcap = softmax_scale / softcap;
        params.scale_softmax = softcap;
        params.scale_softmax_log2 = softcap * M_LOG2E;
    } else{
        // Remove potential NaN
        params.softcap = 0.0;
        params.scale_softmax = softmax_scale;
        params.scale_softmax_log2 = softmax_scale * M_LOG2E;
    }

    // Set this to probability of keeping an element to simplify things.
    params.p_dropout = 1.f - p_dropout;
    // Convert p from float to int so we don't have to convert the random uint to float to compare.
    // [Minor] We want to round down since when we do the comparison we use <= instead of <
    // params.p_dropout_in_uint = uint32_t(std::floor(params.p_dropout * 4294967295.0));
    // params.p_dropout_in_uint16_t = uint16_t(std::floor(params.p_dropout * 65535.0));
    params.p_dropout_in_uint8_t = uint8_t(std::floor(params.p_dropout * 255.0));
    params.rp_dropout = 1.f / params.p_dropout;
    params.scale_softmax_rp_dropout = params.rp_dropout * params.scale_softmax;
    TORCH_CHECK(p_dropout < 1.f);
    #ifdef FLASHATTENTION_DISABLE_DROPOUT
        TORCH_CHECK(p_dropout == 0.0f, "This flash attention build does not support dropout.");
    #endif

    // Causal is the special case where window_size_right == 0 and window_size_left < 0.
    // Local is the more general case where window_size_right >= 0 or window_size_left >= 0.
    params.is_causal = window_size_left < 0 && window_size_right == 0;

    if (window_size_left < 0 && window_size_right >= 0) { window_size_left = seqlen_k; }
    if (window_size_left >= 0 && window_size_right < 0) { window_size_right = seqlen_k; }
    params.window_size_left = window_size_left;
    params.window_size_right = window_size_right;

    #ifdef FLASHATTENTION_DISABLE_LOCAL
        TORCH_CHECK(params.is_causal || (window_size_left < 0 && window_size_right < 0),
            "This flash attention build does not support local attention.");
    #endif

    params.is_seqlens_k_cumulative = true;

    #ifdef FLASHATTENTION_DISABLE_UNEVEN_K
        TORCH_CHECK(d == d_rounded, "This flash attention build does not support headdim not being a multiple of 32.");
    #endif

    params.unpadded_lse = unpadded_lse;
    params.seqlenq_ngroups_swapped = seqlenq_ngroups_swapped;
}

void run_mha_fwd(Flash_fwd_params &params, hipStream_t stream, bool force_split_kernel=false) {
    // For compilation speed, we only build implementations with fp16 datatype and causal attention
    assert(!params.is_bf16);
    assert(params.is_causal);
    // FP16_SWITCH(!params.is_bf16, [&] {
        HEADDIM_SWITCH(params.d, [&] {
            // BOOL_SWITCH(params.is_causal, Is_causal, [&] {
                if (params.num_splits <= 1 && !force_split_kernel) {  // If we don't set it num_splits == 0
                    run_mha_fwd_<cutlass::half_t, kHeadDim, true>(params, stream);
                } else {
                    run_mha_fwd_splitkv_dispatch<cutlass::half_t, kHeadDim, true>(params, stream);
                }
            // });
        });
    // });
}

// Find the number of splits that maximizes the occupancy. For example, if we have
// batch * n_heads = 48 and we have 108 SMs, having 2 splits (efficiency = 0.89) is
// better than having 3 splits (efficiency = 0.67). However, we also don't want too many
// splits as that would incur more HBM reads/writes.
// So we find the best efficiency, then find the smallest number of splits that gets 85%
// of the best efficiency.
inline int num_splits_heuristic(int batch_nheads_mblocks, int num_SMs, int num_n_blocks, int max_splits) {
    // If we have enough to almost fill the SMs, then just use 1 split
    if (batch_nheads_mblocks >= 0.8f * num_SMs) { return 1; }
    max_splits = std::min({max_splits, num_SMs, num_n_blocks});
    float max_efficiency = 0.f;
    std::vector<float> efficiency;
    efficiency.reserve(max_splits);
    auto ceildiv = [](int a, int b) { return (a + b - 1) / b; };
    // Some splits are not eligible. For example, if we have 64 blocks and choose 11 splits,
    // we'll have 6 * 10 + 4 blocks. If we choose 12 splits, we'll have 6 * 11 + (-2) blocks
    // (i.e. it's 11 splits anyway).
    // So we check if the number of blocks per split is the same as the previous num_splits.
    auto is_split_eligible = [&ceildiv, &num_n_blocks](int num_splits) {
        return num_splits == 1 || ceildiv(num_n_blocks, num_splits) != ceildiv(num_n_blocks, num_splits - 1);
    };
    for (int num_splits = 1; num_splits <= max_splits; num_splits++) {
        if (!is_split_eligible(num_splits)) {
            efficiency.push_back(0.f);
        } else {
            float n_waves = float(batch_nheads_mblocks * num_splits) / num_SMs;
            float eff = n_waves / ceil(n_waves);
            // printf("num_splits = %d, eff = %f\n", num_splits, eff);
            if (eff > max_efficiency) { max_efficiency = eff; }
            efficiency.push_back(eff);
        }
    }
    for (int num_splits = 1; num_splits <= max_splits; num_splits++) {
        if (!is_split_eligible(num_splits)) { continue; }
        if (efficiency[num_splits - 1] >= 0.85 * max_efficiency) {
            // printf("num_splits chosen = %d\n", num_splits);
            return num_splits;
        }
    }
    return 1;
}

std::tuple<at::Tensor, at::Tensor> set_params_splitkv(Flash_fwd_params &params, const int batch_size,
    const int num_heads, const int head_size, const int max_seqlen_k, const int max_seqlen_q,
    const int head_size_rounded, const float p_dropout,
    const int num_splits, const int num_sm, struct c10::TensorOptions opts) {

    // This needs to match with run_mha_fwd_splitkv_dispatch
    const int block_n = head_size <= 64 ? 256 : (head_size <= 128 ? 128 : 64);
    const int num_n_blocks = (max_seqlen_k + block_n - 1) / block_n;
    // Technically kBlockM = 64 only for the splitKV kernels, not the standard kernel.
    // In any case we don't expect seqlen_q to be larger than 64 for inference.
    const int num_m_blocks = (max_seqlen_q + 64 - 1) / 64;
    params.num_splits = num_splits;
    at::Tensor softmax_lse_accum;
    at::Tensor out_accum;

    if (p_dropout == 0.0f) {  // SplitKV is not implemented for dropout
        if (num_splits < 1) {
            // We multiply number of SMs by 2 to hard-code the fact that we're using 128 threads per block.
            params.num_splits = num_splits_heuristic(batch_size * num_heads * num_m_blocks, num_sm * 2, num_n_blocks, 128);
        }
        if (params.num_splits > 1) {
            softmax_lse_accum = torch::empty({params.num_splits, batch_size, num_heads, max_seqlen_q}, opts.dtype(at::kFloat));
            out_accum = torch::empty({params.num_splits, batch_size, num_heads, max_seqlen_q, head_size_rounded}, opts.dtype(at::kFloat));
            params.softmax_lseaccum_ptr = softmax_lse_accum.data_ptr();
            params.oaccum_ptr = out_accum.data_ptr();
        }
        TORCH_CHECK(params.num_splits <= 128, "num_splits > 128 not supported");
    }

    return std::make_tuple(softmax_lse_accum, out_accum);
}

void set_params_alibi(Flash_fwd_params &params, const c10::optional<at::Tensor> &alibi_slopes_, int batch_size, int num_heads){
#ifdef FLASHATTENTION_DISABLE_ALIBI
    TORCH_CHECK(!alibi_slopes_.has_value(), "This flash attention build does not support alibi.");
    params.alibi_slopes_ptr = nullptr;
#else
    if (alibi_slopes_.has_value()) {
        auto alibi_slopes = alibi_slopes_.value();
        TORCH_CHECK(alibi_slopes.dtype() == torch::kFloat32, "ALiBi slopes must have dtype fp32");
        CHECK_DEVICE(alibi_slopes);
        TORCH_CHECK(alibi_slopes.stride(-1) == 1, "ALiBi slopes tensor must have contiguous last dimension");
        TORCH_CHECK(alibi_slopes.sizes() == torch::IntArrayRef({num_heads}) || alibi_slopes.sizes() == torch::IntArrayRef({batch_size, num_heads}));
        params.alibi_slopes_ptr = alibi_slopes.data_ptr();
        params.alibi_slopes_batch_stride = alibi_slopes.dim() == 2 ? alibi_slopes.stride(0) : 0;
    } else {
        params.alibi_slopes_ptr = nullptr;
    }
#endif
}

std::vector<at::Tensor>
mha_varlen_fwd(at::Tensor &q,  // total_q x num_heads x head_size, total_q := \sum_{i=0}^{b} s_i
               const at::Tensor &k,  // total_k x num_heads_k x head_size, total_k := \sum_{i=0}^{b} s_i or num_blocks x page_block_size x num_heads_k x head_size if there's a block_table.
               const at::Tensor &v,  // total_k x num_heads_k x head_size, total_k := \sum_{i=0}^{b} s_i or num_blocks x page_block_size x num_heads_k x head_size if there's a block_table.
               const c10::optional<at::Tensor> &out_, // total_q x num_heads x head_size, total_k := \sum_{i=0}^{b} s_i
               const at::Tensor &cu_seqlens_q,  // b+1
               const at::Tensor &cu_seqlens_k,  // b+1
               const c10::optional<at::Tensor> &seqused_k, // b. If given, only this many elements of each batch element's keys are used.
               const c10::optional<const at::Tensor> &leftpad_k_, // batch_size
               const c10::optional<at::Tensor> &block_table_, // batch_size x max_num_blocks_per_seq
               const c10::optional<at::Tensor> &alibi_slopes_, // num_heads or b x num_heads
               int max_seqlen_q,
               const int max_seqlen_k,
               const float p_dropout,
               const float softmax_scale,
               const bool zero_tensors,
               bool is_causal,
               int window_size_left,
               int window_size_right,
               const float softcap,
               const bool return_softmax,
               c10::optional<at::Generator> gen_) {

    // Otherwise the kernel will be launched from cuda:0 device
    at::cuda::CUDAGuard device_guard{q.device()};

    auto [cc_major, cc_minor] = get_compute_capability(get_current_device());
    bool is_sm8x_min = cc_major >= 8;
    TORCH_CHECK(is_sm8x_min, "FlashAttention only supports Ampere GPUs or newer.");

    auto q_dtype = q.dtype();
    TORCH_CHECK(q_dtype == torch::kFloat16 || q_dtype == torch::kBFloat16,
                "FlashAttention only support fp16 and bf16 data type");
    TORCH_CHECK(k.dtype() == q_dtype, "query and key must have the same dtype");
    TORCH_CHECK(v.dtype() == q_dtype, "query and value must have the same dtype");
    TORCH_CHECK(cu_seqlens_q.dtype() == torch::kInt32, "cu_seqlens_q must have dtype int32");
    TORCH_CHECK(cu_seqlens_k.dtype() == torch::kInt32, "cu_seqlens_k must have dtype int32");

    CHECK_DEVICE(q); CHECK_DEVICE(k); CHECK_DEVICE(v);
    CHECK_DEVICE(cu_seqlens_q);
    CHECK_DEVICE(cu_seqlens_k);

    at::Tensor block_table;
    const bool paged_KV = block_table_.has_value();
    if (paged_KV) {
        block_table = block_table_.value();
        CHECK_DEVICE(block_table);
        TORCH_CHECK(block_table.dtype() == torch::kInt32, "block_table must have dtype torch.int32");
        TORCH_CHECK(block_table.stride(-1) == 1, "block_table must have contiguous last dimension");
    }

    TORCH_CHECK(q.stride(-1) == 1, "Input tensor must have contiguous last dimension");
    TORCH_CHECK(k.stride(-1) == 1, "Input tensor must have contiguous last dimension");
    TORCH_CHECK(v.stride(-1) == 1, "Input tensor must have contiguous last dimension");
    CHECK_CONTIGUOUS(cu_seqlens_q);
    CHECK_CONTIGUOUS(cu_seqlens_k);

    const auto sizes = q.sizes();

    const int batch_size = cu_seqlens_q.numel() - 1;
    int num_heads = sizes[1];
    const int head_size = sizes[2];
    const int num_heads_k = paged_KV ? k.size(2) : k.size(1);

    if (softcap > 0.f) { TORCH_CHECK(p_dropout == 0.f, "Softcapping does not support dropout for now"); }

    const int max_num_blocks_per_seq = !paged_KV ? 0 : block_table.size(1);
    const int num_blocks = !paged_KV ? 0 : k.size(0);
    const int page_block_size = !paged_KV ? 1 : k.size(1);
    TORCH_CHECK(!paged_KV || page_block_size % 256 == 0, "Paged KV cache block size must be divisible by 256");

    if (max_seqlen_q == 1 && !alibi_slopes_.has_value()) { is_causal = false; }  // causal=true is the same as causal=false in this case
    if (is_causal) { window_size_right = 0; }

    void *cu_seqlens_q_d = cu_seqlens_q.data_ptr();

    // Faster to transpose q from (b, 1, (nheads_kv ngroups), d) to (b, ngroups, nheads_kv, d) in this case
    // H/t Daniel Haziza
    const int seqlenq_ngroups_swapped = max_seqlen_q == 1 && num_heads > num_heads_k && window_size_left < 0 && window_size_right < 0 && p_dropout == 0.f && head_size % 8 == 0 && !alibi_slopes_.has_value();
    const int ngroups = num_heads / num_heads_k;
    if (seqlenq_ngroups_swapped) {
        q = q.reshape({batch_size, num_heads_k, ngroups, head_size}).transpose(1, 2).reshape({batch_size * ngroups, num_heads_k, head_size});
        max_seqlen_q = ngroups;
        num_heads = num_heads_k;
        cu_seqlens_q_d = nullptr;
    }

    const int total_q = q.sizes()[0];

    TORCH_CHECK(batch_size > 0, "batch size must be positive");
    TORCH_CHECK(head_size <= 256, "FlashAttention forward only supports head dimension at most 256");
    TORCH_CHECK(head_size % 8 == 0, "query, key, value, and out_ must have a head_size that is a multiple of 8");
    TORCH_CHECK(num_heads % num_heads_k == 0, "Number of heads in key/value must divide number of heads in query");

    if (window_size_left >= max_seqlen_k) { window_size_left = -1; }
    if (window_size_right >= max_seqlen_k) { window_size_right = -1; }

    CHECK_SHAPE(q, total_q, num_heads, head_size);
    if (!paged_KV) {
        const int total_k = k.size(0);
        CHECK_SHAPE(k, total_k, num_heads_k, head_size);
        CHECK_SHAPE(v, total_k, num_heads_k, head_size);
    } else {
        CHECK_SHAPE(k, num_blocks, page_block_size, num_heads_k, head_size);
        CHECK_SHAPE(v, num_blocks, page_block_size, num_heads_k, head_size);
        CHECK_SHAPE(block_table, batch_size, max_num_blocks_per_seq);
    }

    CHECK_SHAPE(cu_seqlens_q, batch_size + 1);
    CHECK_SHAPE(cu_seqlens_k, batch_size + 1);
    if (seqused_k.has_value()){
        auto seqused_k_ = seqused_k.value();
        TORCH_CHECK(seqused_k_.dtype() == torch::kInt32, "seqused_k must have dtype int32");
        TORCH_CHECK(seqused_k_.is_cuda(), "seqused_k must be on CUDA device");
        TORCH_CHECK(seqused_k_.is_contiguous(), "seqused_k must be contiguous");
        CHECK_SHAPE(seqused_k_, batch_size);
    }

    at::Tensor out;
    if (out_.has_value()) {
        out = out_.value();
        TORCH_CHECK(out.dtype() == q_dtype, "Output must have the same dtype as inputs");
        CHECK_DEVICE(out);
        TORCH_CHECK(out.stride(-1) == 1, "Output tensor must have contiguous last dimension");
        CHECK_SHAPE(out, sizes[0], sizes[1], head_size);
        if (seqlenq_ngroups_swapped) {
            out = out.reshape({batch_size, num_heads_k, ngroups, head_size}).transpose(1, 2).reshape({batch_size * ngroups, num_heads_k, head_size});
        }
    } else {
        out = torch::empty_like(q);
    }

    auto round_multiple = [](int x, int m) { return (x + m - 1) / m * m; };
    const int head_size_rounded = head_size <= 192 ? round_multiple(head_size, 32) : 256;
    const int seqlen_q_rounded = round_multiple(max_seqlen_q, 128);
    const int seqlen_k_rounded = round_multiple(max_seqlen_k, 128);

    auto opts = q.options();
    auto softmax_lse = torch::empty({num_heads, total_q}, opts.dtype(at::kFloat));
    at::Tensor p;
    // Only return softmax if there's dropout to reduce compilation time
    if (return_softmax) {
        TORCH_CHECK(p_dropout > 0.0f, "return_softmax is only supported when p_dropout > 0.0");
        p = torch::empty({ batch_size, num_heads, seqlen_q_rounded, seqlen_k_rounded }, opts);
    }
    else {
        p = torch::empty({ 0 }, opts);
    }

    if (zero_tensors) {
        out.zero_();
        softmax_lse.fill_(-std::numeric_limits<float>::infinity());
        if (return_softmax) {p.zero_();}
    }

    Flash_fwd_params params;
    set_params_fprop(params,
                     batch_size,
                     max_seqlen_q, max_seqlen_k,
                     seqlen_q_rounded, seqlen_k_rounded,
                     num_heads, num_heads_k,
                     head_size, head_size_rounded,
                     q, k, v, out,
                     cu_seqlens_q_d,
                     cu_seqlens_k.data_ptr(),
                     seqused_k.has_value() ? seqused_k.value().data_ptr() : nullptr,
                     return_softmax ? p.data_ptr() : nullptr,
                     softmax_lse.data_ptr(),
                     p_dropout,
                     softmax_scale,
                     window_size_left,
                     window_size_right,
                     softcap,
                     seqlenq_ngroups_swapped,
                     /*unpadded_lse*/true);
    params.total_q = total_q;

    if (paged_KV) {
        params.block_table = block_table.data_ptr<int>();
        params.block_table_batch_stride = block_table.stride(0);
        params.k_batch_stride = k.stride(0);
        params.v_batch_stride = v.stride(0);
    }
    params.page_block_size = page_block_size;
    // Keep references to these tensors to extend their lifetime
    at::Tensor softmax_lse_accum, out_accum;
    if (seqlenq_ngroups_swapped) {
        // Only apply split-k for decoding
        std::tie(softmax_lse_accum, out_accum) =
            set_params_splitkv(params, batch_size, num_heads, head_size,
                               max_seqlen_k, max_seqlen_q, head_size_rounded,
                               p_dropout, /*num_splits*/ 0, get_num_sm(get_current_device()), opts);
    }

    if (leftpad_k_.has_value()) {
        auto leftpad_k = leftpad_k_.value();
        TORCH_CHECK(!paged_KV, "We don't support Paged KV and leftpad_k running at the same time yet");
        TORCH_CHECK(leftpad_k.dtype() == torch::kInt32, "leftpad_k must have dtype int32");
        CHECK_DEVICE(leftpad_k);
        CHECK_CONTIGUOUS(leftpad_k);
        CHECK_SHAPE(leftpad_k, batch_size);
        params.leftpad_k = static_cast<int *>(leftpad_k.data_ptr());
    }

    // number of times random will be generated per thread, to offset philox counter in thc random
    // state
    // We use a custom RNG that increases the offset by batch_size * nheads * 32.
    int64_t counter_offset = params.b * params.h * 32;
    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA);
    auto rng_state = torch::empty({2}, options.dtype(torch::kInt64));
    // Forward kernel will populate memory with the seed and offset.
    params.rng_state = reinterpret_cast<uint64_t*>(rng_state.data_ptr());

    if (p_dropout > 0.0)  {
        auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(
            gen_, at::cuda::detail::getDefaultCUDAGenerator());
        // See Note [Acquire lock when using random generators]
        std::lock_guard<std::mutex> lock(gen->mutex_);
        params.philox_args = gen->philox_cuda_state(counter_offset);
    }

    set_params_alibi(params, alibi_slopes_, batch_size, num_heads);

    if (max_seqlen_k > 0) {
        auto stream = at::cuda::getCurrentCUDAStream().stream();
        run_mha_fwd(params, stream, paged_KV);
    } else {
        // If seqlen_k == 0, then we have an empty tensor. We need to set the output to 0.
        out.zero_();
        softmax_lse.fill_(std::numeric_limits<float>::infinity());
    }

    if (seqlenq_ngroups_swapped) {
        int64_t size_before[] = {batch_size, max_seqlen_q, num_heads_k, head_size};
        int64_t size_after[] = {batch_size, num_heads_k * max_seqlen_q, head_size};
        out = out.reshape(size_before).transpose(1, 2).reshape(size_after);
        q = q.reshape(size_before).transpose(1, 2).reshape(size_after);
        softmax_lse = softmax_lse.reshape({num_heads * max_seqlen_q, batch_size});
    }

    return {out, softmax_lse, p, rng_state};
}

}   // namespace flash_attention

namespace st::kernel {

void flashAttentionContextStageAttention(
    half* __restrict__ result,     // [num_tokens, local_q_head_num, head_dim]
    const half* __restrict__ qkvs, // [num_tokens, local_q_head_num+2*local_kv_head_num, head_dim]
    const int32_t* __restrict__ ith_context_req_token_index, // [num_context_reqs+1]
    const float qk_scale,
    const int num_q_heads,
    const int num_kv_heads,
    const int head_dim,
    const int num_context_reqs,
    const int max_context_req_len,
    const int num_context_stage_tokens
) {
    auto getTensor = [](void* data, torch::IntArrayRef sizes, const std::vector<int64_t> &strides, torch::ScalarType dtype, torch::Device device = torch::kCUDA) {
        auto options = torch::TensorOptions().dtype(dtype).device(device);
        return torch::from_blob(data, sizes, strides, [](void*) {}, options);
    };
    at::Tensor q_tensor = getTensor(
        const_cast<half*>(qkvs),
        { num_context_stage_tokens, num_q_heads, head_dim },
        { (num_q_heads+2*num_kv_heads)*head_dim, head_dim, 1 },
        torch::kHalf
    );
    at::Tensor k_tensor = getTensor(
        const_cast<half*>(qkvs) + num_q_heads * head_dim,
        { num_context_stage_tokens, num_kv_heads, head_dim },
        { (num_q_heads+2*num_kv_heads)*head_dim, head_dim, 1 },
        torch::kHalf
    );
    at::Tensor v_tensor = getTensor(
        const_cast<half*>(qkvs) + (num_q_heads+num_kv_heads) * head_dim,
        { num_context_stage_tokens, num_kv_heads, head_dim },
        { (num_q_heads+2*num_kv_heads)*head_dim, head_dim, 1 },
        torch::kHalf
    );
    at::Tensor seqstart = getTensor(
        const_cast<int32_t*>(ith_context_req_token_index),
        { num_context_reqs + 1 },
        { 1 },
        torch::kInt32
    );
    at::Tensor result_tensor = getTensor(
        result,
        { num_context_stage_tokens, num_q_heads, head_dim },
        { num_q_heads*head_dim, head_dim, 1 },
        torch::kHalf
    );

    flash_attention::mha_varlen_fwd(
        q_tensor,
        k_tensor,
        v_tensor,
        result_tensor,
        seqstart,
        seqstart,
        c10::nullopt,
        c10::nullopt,
        c10::nullopt,
        c10::nullopt,
        max_context_req_len,
        max_context_req_len,
        0.0f,
        qk_scale,
        false,
        true,
        -1,
        -1,
        0.0f,
        false,
        c10::nullopt
    );  

    sync_check_cuda_error();
}


}	// namespace st::kernel
